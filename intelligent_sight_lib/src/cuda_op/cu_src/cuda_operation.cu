#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cfloat>
#include <cstdint>
#include "cuda_op.h"

hipStream_t CUDASTREAM;

__global__ void rgbToTensor(unsigned char *input, float *output, uint32_t width, uint32_t height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height + 160)
    {
        int idx_out = (y * width + x);
        int size_out = width * (height + 160);

        if (y < height + 80 && y >= 80)
        {
            int idx_in = 3 * ((y - 80) * width + x);

            output[idx_out] = input[idx_in] / 255.0f;                    // R
            output[idx_out + size_out] = input[idx_in + 1] / 255.0f;     // G
            output[idx_out + 2 * size_out] = input[idx_in + 2] / 255.0f; // B
        }
        else
        {
            output[idx_out] = 0.5f;                // R
            output[idx_out + size_out] = 0.5f;     // G
            output[idx_out + 2 * size_out] = 0.5f; // B
        }
    }
}

// assume that output is (640, 640), input is (640, 480), padding 80 pixels on top and bottom
uint16_t convert_rgb888_3dtensor(uint8_t *input_buffer, float *output_buffer, uint32_t width, uint32_t height)
{
    dim3 threads_per_block(16, 16);
    dim3 num_blocks(40, 40);
    rgbToTensor<<<num_blocks, threads_per_block>>>(input_buffer, output_buffer, width, height);
    hipDeviceSynchronize();
    return (uint16_t)hipSuccess;
}

uint16_t transfer_host_to_device(uint8_t *host_mem, uint8_t *device_mem, uint32_t size)
{
    check_status(hipMemcpy(device_mem, host_mem, size, hipMemcpyHostToDevice));
    return (uint16_t)hipSuccess;
}

uint16_t transfer_device_to_host(uint8_t *host_mem, uint8_t *device_mem, uint32_t size)
{
    check_status(hipMemcpy(host_mem, device_mem, size, hipMemcpyDeviceToHost));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_malloc(uint32_t size, uint8_t **buffer)
{
    check_status(hipMalloc((void **)buffer, size));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_malloc_managed(uint32_t size, uint8_t **buffer)
{
    check_status(hipMallocManaged((void **)buffer, size));
    return (uint16_t)hipSuccess;
}

uint16_t cuda_free(uint8_t *buffer)
{
    check_status(hipFree(buffer));
    return (uint16_t)hipSuccess;
}

uint16_t init_cuda()
{
    check_status(hipStreamCreate(&CUDASTREAM));
    return (uint16_t)hipSuccess;
}

uint16_t destroy_cuda()
{
    check_status(hipStreamDestroy(CUDASTREAM));
    return (uint16_t)hipSuccess;
}