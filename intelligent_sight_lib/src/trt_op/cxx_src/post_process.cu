#include "hip/hip_runtime.h"
#include "trt.h"
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

// input tensor shape (1, 32, 8400)
// 32: 4(xywh) + 18(class) + 10(kpnt)
// output shape (1, 8400, 16)
// 16: 4(xywh) + 1(score) + 1(cls) + 10(kpnt)
__global__ void transform_results(float *input_buffer, float *output_buffer)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 8400)
    {
        if (y == 0)
        {
            for (int i = 0; i < 4; i++)
            {
                output_buffer[x * 16 + i] = input_buffer[i * 8400 + x];
            }
            for (int i = 22; i < 32; i++)
            {
                output_buffer[x * 16 + i - 16] = input_buffer[i * 8400 + x];
            }
        }
        else if (y == 1)
        {
            float max_score = input_buffer[4 * 8400 + x];
            int cls = 0;
            for (int i = 5; i < 22; i++)
            {
                if (input_buffer[i * 8400 + x] > max_score)
                {
                    max_score = input_buffer[i * 8400 + x];
                    cls = i - 4;
                }
            }
            output_buffer[x * 16 + 4] = max_score;
            output_buffer[x * 16 + 5] = (float)cls;
        }
    }
}

uint16_t PostProcess::init()
{
    check_status(hipMalloc(&this->transformed, 8400 * 16 * sizeof(float)));
    check_status(hipMalloc(&this->indices, 8400 * sizeof(int)));

    this->d_transformed = thrust::device_ptr<float>(this->transformed);
    this->d_indices = thrust::device_ptr<int>(this->indices);

    check_status(hipHostMalloc(&this->host_transformed, 8400 * 16 * sizeof(float)));
    check_status(hipHostMalloc(&this->host_indices, 8400 * sizeof(int)));

    return (uint16_t)hipSuccess;
}

uint16_t PostProcess::uninit()
{
    check_status(hipFree(this->transformed));
    check_status(hipFree(this->indices));

    check_status(hipHostFree(this->host_transformed));
    check_status(hipHostFree(this->host_indices));

    return (uint16_t)hipSuccess;
}

bool PostProcess::check_iou(float *box1, float *box2)
{
    float x1 = box1[0];
    float y1 = box1[1];
    float w1 = box1[2];
    float h1 = box1[3];
    float x2 = box2[0];
    float y2 = box2[1];
    float w2 = box2[2];
    float h2 = box2[3];
    float area_inter = fmax(fmin(x1 + w1 / 2, x2 + w2 / 2) - fmax(x1 - w1 / 2, x2 - w2 / 2), 0.0f) * fmax(fmin(y1 + h1 / 2, y2 + h2 / 2) - fmax(y1 - h1 / 2, y2 - h2 / 2), 0.0f);
    float area_union = w1 * h1 + w2 * h2 - area_inter;
    return area_inter / area_union > IOU_THRESHOLD;
}

// uint16_t PostProcess::post_process(float *input_buffer, float *output_buffer, uint16_t *num_detections)
// {
//     auto start = std::chrono::high_resolution_clock::now();
//     dim3 threads_pre_block(48, 2);
//     dim3 blocks(175);
//     transform_results<<<blocks, threads_pre_block>>>(input_buffer, this->transformed);
//     check_status(hipDeviceSynchronize());
//     auto end = std::chrono::high_resolution_clock::now();
//     auto diff = end - start;
//     std::cout << "Time taken by 1" << ": " << diff.count() << " seconds" << std::endl;
//     start = std::chrono::high_resolution_clock::now();
//     thrust::sequence(this->d_indices, this->d_indices + 8400);
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 2" << ": " << diff.count() << " seconds" << std::endl;
//     start = std::chrono::high_resolution_clock::now();
//     thrust::sort(this->d_indices, this->d_indices + 8400, [d_transformed = this->d_transformed] __device__(int a, int b)
//                  { return d_transformed[a * 16 + 4] > d_transformed[b * 16 + 4]; });
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 3" << ": " << diff.count() << " seconds" << std::endl;
//     start = std::chrono::high_resolution_clock::now();
//     check_status(hipMemcpy(this->host_indices, this->indices, 8400 * sizeof(int), hipMemcpyDeviceToHost));
//     check_status(hipMemcpy(this->host_transformed, this->transformed, 8400 * 16 * sizeof(float), hipMemcpyDeviceToHost));
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 4" << ": " << diff.count() << " seconds" << std::endl;
//     *num_detections = (uint16_t)MAX_DETECT;
//     start = std::chrono::high_resolution_clock::now();
//     int last = 8400;
//     for (int i = 0; i < 8400; ++i)
//     {
//         if (this->host_transformed[i * 16 + 4] < CONF_THRESHOLD)
//         {
//             last = i;
//             break;
//         }
//     }
//     for (int i = 0, j = 0; i < MAX_DETECT && j != -1; ++i)
//     {
//         int idx = this->host_indices[j];
//         if (this->host_transformed[idx * 16 + 4] < CONF_THRESHOLD)
//         {
//             *num_detections = (uint16_t)i;
//             break;
//         }
//         for (int item = 0; item < 16; ++item)
//         {
//             output_buffer[i * 16 + item] = this->host_transformed[idx * 16 + item];
//         }
//         int next = -1;
//         float *box = this->host_transformed + idx * 16;
//         for (; j < last; ++j)
//         {
//             int idx1 = this->host_indices[j];
//             if (idx1 == -1)
//             {
//                 continue;
//             }
//             if (check_iou(box, this->host_transformed + idx1 * 16))
//             {
//                 this->host_indices[j] = -1;
//             }
//             else if (next == -1)
//             {
//                 next = j;
//             }
//         }
//         j = next;
//     }
//     end = std::chrono::high_resolution_clock::now();
//     diff = end - start;
//     std::cout << "Time taken by 5" << ": " << diff.count() << " seconds" << std::endl;
//     return (uint16_t)hipSuccess;
// }

// input buffer (1, 32, 8400)
// output buffer (MAX_DETECTION, 16)
// 16: 4(xywh) + 1(score) + 1(cls) + 10(kpnt)
uint16_t PostProcess::post_process(float *input_buffer, float *output_buffer, uint16_t *num_detections)
{
    dim3 threads_pre_block(48, 2);
    dim3 blocks(175);
    // (1, 32, 8400)
    transform_results<<<blocks, threads_pre_block>>>(input_buffer, this->transformed);
    // (1, 8400, 16)

    check_status(hipDeviceSynchronize());
    thrust::sequence(this->d_indices, this->d_indices + 8400);
    thrust::sort(this->d_indices, this->d_indices + 8400, [d_transformed = this->d_transformed] __device__(int a, int b)
                 { return d_transformed[a * 16 + 4] > d_transformed[b * 16 + 4]; });

    check_status(hipMemcpy(this->host_indices, this->indices, 8400 * sizeof(int), hipMemcpyDeviceToHost));
    check_status(hipMemcpy(this->host_transformed, this->transformed, 8400 * 16 * sizeof(float), hipMemcpyDeviceToHost));

    // for (int i = 0; i < 16; ++i)
    // {
    //     printf("%f ", this->host_transformed[i]);
    // }
    // printf("\n");

    int last = 8400;
    for (int i = 0; i < 8400; ++i)
    {
        int idx = this->host_indices[i];
        if (this->host_transformed[idx * 16 + 4] < CONF_THRESHOLD)
        {
            last = i;
            break;
        }
    }

    int i = 0;
    for (int j = 0; i < MAX_DETECT && j != -1; ++i)
    {
        int idx = this->host_indices[j];
        for (int item = 0; item < 16; ++item)
        {
            output_buffer[i * 16 + item] = this->host_transformed[idx * 16 + item];
        }

        int next = -1;
        float *box = this->host_transformed + idx * 16;
        for (; j < last; ++j)
        {
            int idx1 = this->host_indices[j];
            if (idx1 == -1)
            {
                continue;
            }
            if (check_iou(box, this->host_transformed + idx1 * 16))
            {
                this->host_indices[j] = -1;
            }
            else if (next == -1)
            {
                next = j;
            }
        }
        j = next;
    }
    *num_detections = (uint16_t)i;
    return (uint16_t)hipSuccess;
}

PostProcess *POSTPROCESS;

uint16_t postprocess_init()
{
    POSTPROCESS = new PostProcess();
    check_status(POSTPROCESS->init());
    return (uint16_t)hipSuccess;
}

// input buffer (1, 32, 8400)
// output buffer (MAX_DETECTION, 16)
// 16: 4(xywh) + 1(score) + 1(cls) + 10(kpnt)
uint16_t postprocess(float *input_buffer, float *output_buffer, uint16_t *num_detections)
{
    check_status(POSTPROCESS->post_process(input_buffer, output_buffer, num_detections));
    return (uint16_t)hipSuccess;
}

uint16_t postprocess_destroy()
{
    check_status(POSTPROCESS->uninit());
    delete POSTPROCESS;
    return (uint16_t)hipSuccess;
}